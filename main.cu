
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
//#include "initialInnerPlanets.h"
//#include "taylorJorge.h"
//#include "nbodyJorge.h"

int nplanetas, tamano, dimensiones, N, xyz, gmax;
float *d_Gm, *d_aux, *d_q, *d_ddq;
bool gpu;

//#include "nbodyJorge.h"
/*
extern int nplanetas;
extern int xyz;
extern int tamano;
extern int dimensiones;
extern int N;
extern int gmax;*/

__global__
void NbodyODE2Tpgpuaux(int gmax, int nplanetas, int xyz, int tamano, int dimensiones, int N, float *Gm, float *aux, float *q, float *ddq) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    //int i = threadIdx.x;
    //int j = threadIdx.y;

    float Gmi, Gmj;
    //float Gmi;

    if (i < nplanetas && j < nplanetas && i < j) {
    //if (i < nplanetas) {
    //for (int i=0; i<nplanetas; i++) {
        Gmi = Gm[i];
        //for (int j=i+1; j<nplanetas; j++) {
        //   float Gmj = Gm[j];
        //Gmi = Gm[i];
        Gmj = Gm[j];
        
            for (int k=0; k<dimensiones; k++) {
                aux[k*N+0] = q[k*tamano+0+i*xyz] - q[k*tamano+0+j*xyz];
                aux[k*N+1] = q[k*tamano+1+i*xyz] - q[k*tamano+1+j*xyz];
                aux[k*N+2] = q[k*tamano+2+i*xyz] - q[k*tamano+2+j*xyz];
            }
            for (int k=0; k<dimensiones; k++) {
                aux[k*N+3] = 0.0;
                aux[k*N+4] = 0.0;
                aux[k*N+5] = 0.0;
                for (int m=0; m<=k; m++) {
                    aux[k*N+3] += aux[m*N+0] * aux[(k-m)*N+0];
                    aux[k*N+4] += aux[m*N+1] * aux[(k-m)*N+1];
                    aux[k*N+5] += aux[m*N+2] * aux[(k-m)*N+2];
                }
            }

            for (int k=0; k<dimensiones; k++) {
                aux[k*N+6] = aux[k*N+3] + aux[k*N+4] + aux[k*N+5];
                aux[k*N+7] = 0.0;
            }

            float berretzailea = -3.0/2.0;
            aux[7] = pow(aux[6],berretzailea);
            for (int k=1; k<=dimensiones-1; k++) {
                float lag = 0.0;
                for (int m=0; m<=k; m++) {
                    lag = lag + (berretzailea*(k-m)-m)*aux[(k-1-m+1)*N+6]*aux[m*N+7];
                }
                aux[k*N+7]=lag/(k*aux[6]);
            }            

            for (int k=0; k<dimensiones; k++) {
                aux[k*N+3]=0.0;
                aux[k*N+4]=0.0;
                aux[k*N+5]=0.0;
                for (int m=1; m<=k+1; m++) {
                    aux[k*N+3] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N];
                    aux[k*N+4] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N+1];
                    aux[k*N+5] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N+2];
                }
            }

            for (int k=0; k<dimensiones; k++) { 
                ddq[k*tamano+0+i*xyz] -= Gmj*aux[k*N+3];
                ddq[k*tamano+0+j*xyz] += Gmi*aux[k*N+3];
                ddq[k*tamano+1+i*xyz] -= Gmj*aux[k*N+4];
                ddq[k*tamano+1+j*xyz] += Gmi*aux[k*N+4];
                ddq[k*tamano+2+i*xyz] -= Gmj*aux[k*N+5];
                ddq[k*tamano+2+j*xyz] += Gmi*aux[k*N+5];
            }
        }
    //}
    
}

void NbodyODE2Tpgpu(float *ddq, float *q, float *Gm, float *aux) {
    
    for (int i=0; i<tamano*gmax; i++) {
        ddq[i] = 0.0;
    }
    
    /*cudaMemcpy(d_Gm, Gm, nplanetas*sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(d_aux, aux, N*xyz*sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(d_q, q, tamano*gmax*sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(d_ddq, ddq, tamano*gmax*sizeof(float), cudaMemcpyHostToDevice);*/

    //int threadsPerBlock = 256;
    //int blocksPerGrid = (nplanetas + threadsPerBlock - 1) / threadsPerBlock;
    
    NbodyODE2Tpgpuaux<<<(xyz*tamano*dimensiones)/32, 32>>>(gmax, nplanetas, xyz, tamano, dimensiones, N, d_Gm, d_aux, d_q, d_ddq);

    /*cudaMemcpy(Gm, d_Gm, nplanetas*sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(aux, d_aux, N*xyz*sizeof(float), cudaMemcpyDeviceToHost);
    cudaMemcpy(q, d_q, tamano*gmax*sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(ddq, d_ddq, tamano*gmax*sizeof(float), cudaMemcpyDeviceToHost);*/
}

void inicializarGlobales() {
    nplanetas = 5;
    xyz = 3;
    tamano = nplanetas*xyz;
    dimensiones = 2;
    N = 8;
    gmax = 8;
    gpu = true;
}

void NbodyODE2Tp(float *ddq, float *q, float *Gm, float *aux) {

    for (int i=0; i<tamano*gmax; i++) {
        ddq[i] = 0.0;
    }
    
    for (int i=0; i<nplanetas; i++) {
        float Gmi = Gm[i];
        for (int j=i+1; j<nplanetas; j++) {
           float Gmj = Gm[j];
           for (int k=0; k<dimensiones; k++) {
               aux[k*N+0] = q[k*tamano+0+i*xyz] - q[k*tamano+0+j*xyz];
               aux[k*N+1] = q[k*tamano+1+i*xyz] - q[k*tamano+1+j*xyz];
               aux[k*N+2] = q[k*tamano+2+i*xyz] - q[k*tamano+2+j*xyz];
           }
           for (int k=0; k<dimensiones; k++) {
                aux[k*N+3] = 0.0;
                aux[k*N+4] = 0.0;
                aux[k*N+5] = 0.0;
                for (int m=0; m<=k; m++) {
                    aux[k*N+3] += aux[m*N+0] * aux[(k-m)*N+0];
                    aux[k*N+4] += aux[m*N+1] * aux[(k-m)*N+1];
                    aux[k*N+5] += aux[m*N+2] * aux[(k-m)*N+2];
                }
           }

            for (int k=0; k<dimensiones; k++) {
                aux[k*N+6] = aux[k*N+3] + aux[k*N+4] + aux[k*N+5];
                aux[k*N+7] = 0.0;
            }


            float berretzailea = -3.0/2.0;
            aux[7] = pow(aux[6],berretzailea);
            for (int k=1; k<=dimensiones-1; k++) {
                float lag = 0.0;
                for (int m=0; m<=k; m++) {
                    lag = lag + (berretzailea*(k-m)-m)*aux[(k-1-m+1)*N+6]*aux[m*N+7];
                }
                aux[k*N+7]=lag/(k*aux[6]);
            }            

            for (int k=0; k<dimensiones; k++) {
                aux[k*N+3]=0.0;
                aux[k*N+4]=0.0;
                aux[k*N+5]=0.0;
                for (int m=1; m<=k+1; m++) {
                    aux[k*N+3] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N];
                    aux[k*N+4] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N+1];
                    aux[k*N+5] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N+2];
                }
            }

            for (int k=0; k<dimensiones; k++) { 
                ddq[k*tamano+0+i*xyz] -= Gmj*aux[k*N+3];
                ddq[k*tamano+0+j*xyz] += Gmi*aux[k*N+3];
                ddq[k*tamano+1+i*xyz] -= Gmj*aux[k*N+4];
                ddq[k*tamano+1+j*xyz] += Gmi*aux[k*N+4];
                ddq[k*tamano+2+i*xyz] -= Gmj*aux[k*N+5];
                ddq[k*tamano+2+j*xyz] += Gmi*aux[k*N+5];
            }
        }
    }
}

void TaylorLortuP(float *nu, float *nddu, float *gm, float *aux) {
    int gradua = gmax-1;
    for (int i = 0; i<gradua/2; i++) {
        if (gpu) {
            NbodyODE2Tpgpu(nddu,nu,gm,aux);
        } else {
            NbodyODE2Tp(nddu,nu,gm,aux);
        }
        
        int k = 2*(i+1)+1;

        if (k < gmax) {   
            float zat=(k-1)*(k-2);
            float zat2=k*(k-1);
            dimensiones+=2;
            for (int body=0; body<nplanetas; body++) {
                for (int koor=0; koor<xyz; koor++) {
                    nu[(k-1)*tamano+koor*nplanetas+body] =nddu[(k-3)*tamano+koor*nplanetas+body]/zat;
                    nu[k*tamano+koor*nplanetas+body] =nddu[(k-2)*tamano+koor*nplanetas+body]/zat2;
                }
            }
        } else {
            float zat=(k-1)*(k-2);
            dimensiones++;
            for (int body=0; body<nplanetas; body++) {
                for (int koor=0; koor<xyz; koor++) {
                    nu[(k-1)*tamano+koor*nplanetas+body] = nddu[(k-3)*tamano+koor*nplanetas+body]/zat;
                }
            }
        }       

    }
}

void evaluatetaylorv(float *nu, float h) {

    float x, y, z, dx, dy, dz, ukx, uky, ukz;

    for (int body=0; body<nplanetas; body++) {//body in 1:N
        x = nu[(dimensiones-1)*tamano+0+body*xyz]*h;
        nu[(dimensiones-1)*tamano+0+body*xyz]=0.0;
        y= nu[(dimensiones-1)*tamano+1+body*xyz]*h;
        nu[(dimensiones-1)*tamano+1+body*xyz]=0.0;
        z= nu[(dimensiones-1)*tamano+2+body*xyz]*h;
        nu[(dimensiones-1)*tamano+2+body*xyz]=0.0;
        dx= (dimensiones-1)*x;
        dy= (dimensiones-1)*y;
        dz= (dimensiones-1)*z;
        for (int k=dimensiones-1; k>1; k--) {//k in n-1:-1:3 // azken bi terminoak era berezian tratatuko ditut,
                          // izan ere, u'(t) espresioak ez baitauka t biderkatzen.
            ukx= nu[k*tamano+0+body*xyz];
            nu[k*tamano+0+body*xyz]=0.0;
            uky= nu[k*tamano+1+body*xyz];
            nu[k*tamano+1+body*xyz]=0.0;
            ukz= nu[k*tamano+2+body*xyz];
            nu[k*tamano+2+body*xyz]=0.0;
            x+=ukx;
            x*=h;
            y+=uky;
            y*=h;
            z+=ukz;
            z*=h;
            dx+= (k-1)*ukx;
            dx*=h;
            dy+= (k-1)*uky;
            dy*=h;
            dz+= (k-1)*ukz;
            dz*=h;
        }
        // orain bigarren terminoari dagokiona: 
        // u(t) kasuan orain artekoa honi gehitu eta dena * h
        // u'(t) kasuan  u_2 balioari gehitu orain artekoa eta laga u_2 berri bezala
        ukx= nu[1*tamano+0+body*xyz];
        uky= nu[1*tamano+1+body*xyz];
        ukz= nu[1*tamano+2+body*xyz];
        x+=ukx;
        x*=h;
        y+=uky;
        y*=h;
        z+=ukz;
        z*=h;
        nu[1*tamano+0+body*xyz] += dx;
        nu[1*tamano+1+body*xyz] += dy;
        nu[1*tamano+2+body*xyz] += dz;
        // Eta bukatzeko, lehenengo terminoari dagokiona: hau u(t) espresioan bakarrik dago,
        // eta u_1 koefizienteari orain artekoa gehitu behar diot
        nu[/*0*tamano+0+*/body*xyz] += x;
        nu[1+body*xyz] += y;
        nu[2+body*xyz] += z;
    }
}

void TaylorStepP(float *u, float *ddu, float *gm, float *aux, float h) {
    TaylorLortuP(u,ddu,gm,aux);
    evaluatetaylorv(u,h);
}


void IntegrateTaylorP(float *u, int t0, int tf, float h, float *gm) {

    float *ddu;
    ddu = (float *) malloc(tamano*gmax*sizeof(float));
    for (int i=0; i<tamano*gmax; i++) {
        ddu[i] = 0.0;
    }

    float *aux;
    aux = (float *) malloc(N*xyz*sizeof(float));
    for (int i=0; i<N*xyz; i++) {
        aux[i] = 0.0;
    }

    printf("Inicializada u:\n");
    for (int i=0; i<gmax; i++){
        for (int j=0; j<nplanetas; j++) {
            printf("%g %g %g;   ", u[i*tamano+j*xyz], u[i*tamano+j*xyz+1], u[i*tamano+j*xyz+2]);
        }
        printf("\n");
    }

    if (gpu) {
        hipMalloc(&d_Gm, nplanetas*sizeof(float));
        hipMalloc(&d_aux, N*xyz*sizeof(float));
        hipMalloc(&d_q, tamano*gmax*sizeof(float));
        hipMalloc(&d_ddq, tamano*gmax*sizeof(float));

        hipMemcpy(d_Gm, gm, nplanetas*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_aux, aux, N*xyz*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_q, u, tamano*gmax*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_ddq, ddu, tamano*gmax*sizeof(float), hipMemcpyHostToDevice);
    }

    int ukop = (tf-t0)/h;
    printf("%d\n", ukop);
    ukop = 100000;
    clock_t begin = clock();
    for (int i=1; i<=ukop; i++) {
        if (gpu) {
            TaylorStepP(u, ddu, gm, aux, h);//TaylorStepP(d_q, d_ddq, d_Gm, d_aux, h);
        } else {
            TaylorStepP(u, ddu, gm, aux, h);
        }
        dimensiones = 2;
    }
    clock_t end = clock();

    if (gpu) {
        /*cudaMemcpy(gm, d_Gm, nplanetas*sizeof(float), cudaMemcpyHostToDevice);
        cudaMemcpy(aux, d_aux, N*xyz*sizeof(float), cudaMemcpyDeviceToHost);
        cudaMemcpy(u, d_q, tamano*gmax*sizeof(float), cudaMemcpyHostToDevice);
        cudaMemcpy(ddu, d_ddq, tamano*gmax*sizeof(float), cudaMemcpyDeviceToHost);*/
    }

    printf("FINAL");
    for (int i=0; i<gmax; i++){
        for (int j=0; j<nplanetas; j++) {
            printf("%g %g %g;   ", u[i*tamano+j*xyz], u[i*tamano+j*xyz+1], u[i*tamano+j*xyz+2]);
        }
        printf("\n");
    }

    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    printf("Tiempo: %f\n", time_spent);
    
    if (gpu) {
        hipFree(d_Gm);
        hipFree(d_aux);
        hipFree(d_q);
        hipFree(d_ddq);
    }
    free(ddu);
    free(aux);
}

void initialInnerPlanets(float *Gm, float *u, int tamano) {

/*
Order of planets:
Sun, Mercury, Venus, Earth, Mars, Moon
*/

    float GmMoon = 0.109318945074237400e-10;
    float GmEarth = 0.888769244512563400e-9;

    Gm[0] = 0.295912208285591100e-3;
    Gm[1] = 0.491248045036476000e-10;
    Gm[2] = 0.724345233264412000e-9;
    Gm[3] = GmEarth+GmMoon;
    Gm[4] = 0.954954869555077000e-10;

    float q0[15] = {0.00450250878464055477, 0.00076707642709100705, 0.00026605791776697764,   // Sun
        0.36176271656028195477, -0.09078197215676599295, -0.08571497256275117236, // Mercury
        0.61275194083507215477, -0.34836536903362219295, -0.19527828667594382236, // Venus
        0.12051741410138465477, -0.92583847476914859295, -0.40154022645315222236, // EM bary
        -0.11018607714879824523, -1.32759945030298299295,-0.60588914048429142236  // Mars
    };

    float v[15] = {-0.00000035174953607552, 0.00000517762640983341, 0.00000222910217891203,  // Sun
      0.00336749397200575848, 0.02489452055768343341, 0.01294630040970409203,  // Mercury
      0.01095206842352823448, 0.01561768426786768341, 0.00633110570297786403,  // Venus
      0.01681126830978379448, 0.00174830923073434441, 0.00075820289738312913,  // EM bary
      0.01448165305704756448, 0.00024246307683646861, -0.00028152072792433877  // Mars
    };

    float sumGm = 0.0;
    for (int i=0; i<5; i++) {
       sumGm = sumGm + Gm[i];
    }

    float resultadoqx = 0.0;
    float resultadoqy = 0.0;
    float resultadoqz = 0.0;
    for (int i=0; i<5; i++) {
       resultadoqx =  resultadoqx + (Gm[i]*q0[i*3]);
       resultadoqy =  resultadoqy + (Gm[i]*q0[i*3+1]);
       resultadoqz =  resultadoqz + (Gm[i]*q0[i*3+2]);
    }
    float qxbar = resultadoqx/sumGm;
    float qybar = resultadoqy/sumGm;
    float qzbar = resultadoqz/sumGm;

    float resultadovx = 0.0;
    float resultadovy = 0.0;
    float resultadovz = 0.0;
    for (int i=0; i<5; i++) {
       resultadovx =  resultadovx + (Gm[i]*v[i*3]);
       resultadovy =  resultadovy + (Gm[i]*v[i*3+1]);
       resultadovz =  resultadovz + (Gm[i]*v[i*3+2]);
    }
    float vxbar = resultadovx/sumGm;
    float vybar = resultadovy/sumGm;
    float vzbar = resultadovz/sumGm;

    for (int i=0; i<5; i++) {
       q0[i*3] = q0[i*3] - qxbar;
       q0[i*3+1] = q0[i*3+1] - qybar;
       q0[i*3+2] = q0[i*3+2] - qzbar;
    }

    for (int i=0; i<5; i++) {
       v[i*3] = v[i*3] - vxbar;
       v[i*3+1] = v[i*3+1] - vybar;
       v[i*3+2] = v[i*3+2] - vzbar;
    }

    for (int i=0; i<tamano; i++) {
        u[i] = q0[i];
        u[i+tamano] = v[i];
    }

    for (int i=nplanetas*xyz*2; i<gmax*nplanetas*xyz; i++) {
        u[i] = 0.0;
    }

}


int main() {

    inicializarGlobales();

    float *GM;
    GM = (float *) malloc(nplanetas*sizeof(float));
    
    float *u;
    u = (float *) malloc(tamano*gmax*sizeof(float));

    initialInnerPlanets(GM, u, tamano);

    int t0 = 0;
    int tf = 360;
    float h = 0.01;

    IntegrateTaylorP(u, t0, tf, h, GM);

    free(GM);
    free(u);
    
}
