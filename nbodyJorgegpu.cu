#include "hip/hip_runtime.h"
#include "nbodyJorge.h"
/*
extern int nplanetas;
extern int xyz;
extern int tamano;
extern int dimensiones;
extern int N;
extern int gmax;*/

__global__
void NbodyODE2Tpgpuaux(int nplanetas, int xyz, int tamano, int dimensiones, int N, float *Gm, float *aux, float *q, float *ddq) {

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = threadIdx.x;
    int j = threadIdx.y;

    float Gmi, Gmj;

    if (i < nplanetas && j < nplanetas && i < j) {
        Gmi = Gm[i];
        Gmj = Gm[j];
        
        for (int k=0; k<dimensiones; k++) {
            aux[k*N+0] = q[k*tamano+0+i*xyz] - q[k*tamano+0+j*xyz];
            aux[k*N+1] = q[k*tamano+1+i*xyz] - q[k*tamano+1+j*xyz];
            aux[k*N+2] = q[k*tamano+2+i*xyz] - q[k*tamano+2+j*xyz];
        }
        for (int k=0; k<dimensiones; k++) {
            aux[k*N+3] = 0.0;
            aux[k*N+4] = 0.0;
            aux[k*N+5] = 0.0;
            for (int m=0; m<=k; m++) {
                aux[k*N+3] += aux[m*N+0] * aux[(k-m)*N+0];
                aux[k*N+4] += aux[m*N+1] * aux[(k-m)*N+1];
                aux[k*N+5] += aux[m*N+2] * aux[(k-m)*N+2];
            }
        }

        for (int k=0; k<dimensiones; k++) {
            aux[k*N+6] = aux[k*N+3] + aux[k*N+4] + aux[k*N+5];
            aux[k*N+7] = 0.0;
        }

        float berretzailea = -3.0/2.0;
        aux[7] = pow(aux[6],berretzailea);
        for (int k=1; k<=dimensiones-1; k++) {
            float lag = 0.0;
            for (int m=0; m<=k; m++) {
                lag = lag + (berretzailea*(k-m)-m)*aux[(k-1-m+1)*N+6]*aux[m*N+7];
            }
            aux[k*N+7]=lag/(k*aux[6]);
        }            

        for (int k=0; k<dimensiones; k++) {
            aux[k*N+3]=0.0;
            aux[k*N+4]=0.0;
            aux[k*N+5]=0.0;
            for (int m=1; m<=k+1; m++) {
                aux[k*N+3] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N];
                aux[k*N+4] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N+1];
                aux[k*N+5] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N+2];
            }
        }

        for (int k=0; k<dimensiones; k++) { 
            ddq[k*tamano+0+i*xyz] -= Gmj*aux[k*N+3];
            ddq[k*tamano+0+j*xyz] += Gmi*aux[k*N+3];
            ddq[k*tamano+1+i*xyz] -= Gmj*aux[k*N+4];
            ddq[k*tamano+1+j*xyz] += Gmi*aux[k*N+4];
            ddq[k*tamano+2+i*xyz] -= Gmj*aux[k*N+5];
            ddq[k*tamano+2+j*xyz] += Gmi*aux[k*N+5];
        }
    }
    
    /*for (int i=0; i<nplanetas; i++) {
        float Gmi = Gm[i];
        for (int j=i+1; j<nplanetas; j++) {
           float Gmj = Gm[j];
           for (int k=0; k<dimensiones; k++) {
               aux[k*N+0] = q[k*tamano+0+i*xyz] - q[k*tamano+0+j*xyz];
               aux[k*N+1] = q[k*tamano+1+i*xyz] - q[k*tamano+1+j*xyz];
               aux[k*N+2] = q[k*tamano+2+i*xyz] - q[k*tamano+2+j*xyz];
           }
           for (int k=0; k<dimensiones; k++) {
                aux[k*N+3] = 0.0;
                aux[k*N+4] = 0.0;
                aux[k*N+5] = 0.0;
                for (int m=0; m<=k; m++) {
                    aux[k*N+3] += aux[m*N+0] * aux[(k-m)*N+0];
                    aux[k*N+4] += aux[m*N+1] * aux[(k-m)*N+1];
                    aux[k*N+5] += aux[m*N+2] * aux[(k-m)*N+2];
                }
           }

            for (int k=0; k<dimensiones; k++) {
                aux[k*N+6] = aux[k*N+3] + aux[k*N+4] + aux[k*N+5];
                aux[k*N+7] = 0.0;
            }


            float berretzailea = -3.0/2.0;
            aux[7] = pow(aux[6],berretzailea);
            for (int k=1; k<=dimensiones-1; k++) {
                float lag = 0.0;
                for (int m=0; m<=k; m++) {
                    lag = lag + (berretzailea*(k-m)-m)*aux[(k-1-m+1)*N+6]*aux[m*N+7];
                }
                aux[k*N+7]=lag/(k*aux[6]);
            }            

            for (int k=0; k<dimensiones; k++) {
                aux[k*N+3]=0.0;
                aux[k*N+4]=0.0;
                aux[k*N+5]=0.0;
                for (int m=1; m<=k+1; m++) {
                    aux[k*N+3] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N];
                    aux[k*N+4] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N+1];
                    aux[k*N+5] +=  aux[(m-1)*N+7]*aux[(k-m+1)*N+2];
                }
            }

            for (int k=0; k<dimensiones; k++) { 
                ddq[k*tamano+0+i*xyz] -= Gmj*aux[k*N+3];
                ddq[k*tamano+0+j*xyz] += Gmi*aux[k*N+3];
                ddq[k*tamano+1+i*xyz] -= Gmj*aux[k*N+4];
                ddq[k*tamano+1+j*xyz] += Gmi*aux[k*N+4];
                ddq[k*tamano+2+i*xyz] -= Gmj*aux[k*N+5];
                ddq[k*tamano+2+j*xyz] += Gmi*aux[k*N+5];
            }
        }
    }*/
}

void NbodyODE2Tpgpu(int nplanetas, int xyz, int tamano, int dimensiones, int N, int gmax, float *ddq, float *q, struct NBodyParams *p) {

    for (int i=0; i<tamano*gmax; i++) {
        ddq[i] = 0.0;
    }
    float *Gm = p->gm;
    float *aux = p->aux;

    float *d_Gm, *d_aux, *d_q, *d_ddq;
    hipMalloc(&d_Gm, nplanetas*sizeof(float));
    hipMalloc(&d_aux, N*xyz*sizeof(float));
    hipMalloc(&d_q, tamano*gmax*sizeof(float));
    hipMalloc(&d_ddq, tamano*gmax*sizeof(float));

    hipMemcpy(d_Gm, Gm, nplanetas*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_aux, aux, N*xyz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_q, q, tamano*gmax*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ddq, ddq, tamano*gmax*sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nplanetas + threadsPerBlock - 1) / threadsPerBlock;
    NbodyODE2Tpgpuaux<<<blocksPerGrid, threadsPerBlock>>>(nplanetas, xyz, tamano, dimensiones, N, d_Gm, d_aux, d_q, d_ddq);

    hipMemcpy(aux, d_aux, N*xyz*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(ddq, d_ddq, tamano*gmax*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_Gm);
    hipFree(d_aux);
    hipFree(d_q);
    hipFree(d_ddq);

}