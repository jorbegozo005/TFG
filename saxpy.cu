#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

__global__
void prueba(int n, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = y[i];
}

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  printf("Bloque = %d, Thread = %d\n", blockIdx.x, threadIdx.x);
  
  if (i < n) {
    y[i] = a*x[i] + y[i];
  }
}

void saxpycpu(int n, float a, float *x, float *y) {
  for (int i = 0; i < n; i++) {
    y[i] = a*x[i] + y[i];
    //printf("Eso %f\n", y[i]);
  }
}

int main(void)
{
  
  int N = 5;//1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    printf("%d ",i);
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  clock_t begin = clock();

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  for (int i=0; i<5; i++) {

    printf("Antes\n");

    //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
    saxpy<<<5,3>>>(N, 2.0f, d_x, d_y);

    hipDeviceSynchronize();
    printf("Sinc\n");
  }
  //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  //hipDeviceSynchronize();

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  clock_t end = clock();
  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

  printf("Tiempo gpu: %f\n", time_spent);

  for (int i = 0; i < N; i++) {
    printf("%d ",i);
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  begin = clock();

  for (int i=0; i<5; i++) {

    saxpycpu(N, 2.0f, x, y);
    
  }

  //saxpycpu(N, 2.0f, x, y);

  end = clock();
  time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

  printf("Tiempo cpu: %f\n", time_spent);


  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);


}