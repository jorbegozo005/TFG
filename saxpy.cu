#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void saxpycpu(int n, float a, float *x, float *y) {
  for (int i = 0; i < n; i++) {
    y[i] = a*x[i] + y[i];
  }
}

int main(void)
{
  clock_t begin = clock();

  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  saxpycpu(N, 2.0f, x, y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  clock_t end = clock();
  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

  printf("Tiempo: %f\n", time_spent);

}